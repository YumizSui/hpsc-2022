
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <vector>

__global__ void prepareBucket(int *key, int *bucket){
  int i = threadIdx.x;
  atomicAdd(&bucket[key[i]], 1);
}

__global__ void countKey(int *key, int *bucket){
  int i = threadIdx.x;
  int j = 0;
  // calc start index j
  for (int k=0; k<i; k++) {
    j += bucket[k];
  }

  // count key
  for (; bucket[i]>0; bucket[i]--) {
    key[j++] = i;
  }
}

int main() {
  int n = 50;
  int range = 5;
  int *key, *bucket;
  hipMallocManaged(&key, n*sizeof(int));
  hipMallocManaged(&bucket, range*sizeof(int));
  for (int i=0; i<n; i++) {
    key[i] = rand() % range;
    printf("%d ",key[i]);
  }
  printf("\n");
  for (int i=0; i<range; i++) {
    bucket[i] = 0;
  }
  
  prepareBucket<<<1,n>>>(key, bucket);
  countKey<<<1,range>>>(key, bucket);

  hipDeviceSynchronize();
  for (int i=0; i<n; i++) {
    printf("%d ",key[i]);
  }
  printf("\n");
  hipFree(key);
  hipFree(bucket);
}